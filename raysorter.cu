
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#include "../SingleBounceKernel/sbk.h"
#include "../cuda-raytracing-optimized/staircase_scene.h"

struct MortonKey
{
    signed int      oldSlot;
    unsigned int    hash[6];        // 192-bit Morton key
};

void computeTmax(saved_path* paths, uint32_t numpaths, bbox bounds);
void computeAABB(const saved_path* paths, uint32_t numpaths, bbox* aabb);
void computeMortonCodes(const saved_path* paths, uint32_t numpaths, MortonKey* keys);
void sortPaths(const saved_path* paths, uint32_t numpaths, const MortonKey* keys, saved_path* sorted);

int main(int argc, char** argv)
{
    int nx = 320;
    int ny = 400;
    int ns = 64;

    // extract bounce number from args
    int bounce = 0;
    if (argc > 1)
        bounce = strtol(argv[1], NULL, 10);
    std::string file = filename(bounce, ns);
    std::cerr << "loading bounce file " << file << std::endl;

    // load rays from bounce file
    uint32_t numpaths = nx * ny * ns;
    saved_path* paths = new saved_path[numpaths];
    if (!load(file, paths, numpaths)) {
        return -1;
    }

    // load scene's bvh and retrieve mesh bounds
    std::cerr << "loading scene" << std::endl;
    mesh* m = new mesh();
    int numPrimitivesPerLeaf = 0;
    if (!loadBVH("D:\\models\\obj\\staircase.bvh", *m, numPrimitivesPerLeaf)) {
        return -1;
    }
    bbox bounds = m->bounds;
    delete m;
    m = NULL;

    // compute tmax for all paths
    std::cerr << "computing tmax" << std::endl;
    computeTmax(paths, numpaths, bounds);

    // compute aabb for all paths
    std::cerr << "computing aabb" << std::endl;
    bbox* aabb = new bbox[numpaths];
    computeAABB(paths, numpaths, aabb);

    // compute Morton code for all rays
    std::cerr << "computing Morton codes" << std::endl;
    MortonKey* keys = new MortonKey[numpaths];
    computeMortonCodes(paths, numpaths, keys);

    // sort rays using their Morton codes
    std::cerr << "sorting paths" << std::endl;
    saved_path* sorted = new saved_path[numpaths];
    sortPaths(paths, numpaths, keys, sorted);

    // save rays to .sorted file
    std::string outfile = file + ".sorted";
    std::cerr << "saving file " << outfile << std::endl;
    save(outfile, sorted, numpaths);

    // cleanup
    delete[] paths;
    delete[] aabb;
    delete[] keys;
    delete[] sorted;

    return 0;
}

void computeTmax(saved_path* paths, uint32_t numpaths, bbox bounds) {
}

void computeAABB(const saved_path* paths, uint32_t numpaths, bbox* aabb) {
}

void computeMortonCodes(const saved_path* paths, uint32_t numpaths, MortonKey* keys) {
}

void sortPaths(const saved_path* paths, uint32_t numpaths, const MortonKey* keys, saved_path* sorted) {
}