#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>

#define DEBUG_SORT

#define SAVE_BITSTACK

#define STB_IMAGE_IMPLEMENTATION
#include "../cuda-raytracing-optimized/stb_image.h"

#include "../SingleBounceKernel/sbk.h"
#include "../cuda-raytracing-optimized/staircase_scene.h"

struct MortonKey
{
    int         oldSlot;
    uint32_t    hash[6];        // 192-bit Morton key
};

struct light_path {
    vec3 origin;
    vec3 direction;
    float tmax;

    light_path() {}
    light_path(const saved_path& p) : origin(p.origin), direction(p.rayDir), tmax(FLT_MAX) {}
};

void computeTmax(light_path* paths, uint32_t numpaths, bbox bounds);
bbox computeAABB(const light_path* paths, uint32_t numpaths);
void computeMortonCodes(const light_path* paths, uint32_t numpaths, const bbox aabb, MortonKey* keys);
void sortPaths(const saved_path* paths, uint32_t numpaths, MortonKey* keys, saved_path* sorted);

bool tmax_bbox(const bbox& bounds, light_path& p);
void collectBits(unsigned int* hash, int index, unsigned int value);
int compareMortonKey(const void* A, const void* B);
int compareSavedPath(const void* A, const void* B);

int main(int argc, char** argv)
{
    int nx = 320;
    int ny = 400;
    int ns = 64;

    // extract bounce number from args
    int bounce = 0;
    if (argc > 1)
        bounce = strtol(argv[1], NULL, 10);
    std::string file = filename(bounce, ns, false);
    std::cerr << "loading bounce file " << file << std::endl;

    // load rays from bounce file
    uint32_t numpaths = nx * ny * ns;
    saved_path* paths = new saved_path[numpaths];
    light_path* light = new light_path[numpaths];
    if (!load(file, paths, numpaths)) {
        return -1;
    }
    //// replace bitstack with a random number
    //std::random_device rd;
    //std::mt19937 gen(rd());
    //for (auto i = 0; i < numpaths; i++) {
    //    paths[i].bitstack = gen();
    //}

    for (auto i = 0; i < numpaths; i++) {
        light[i] = light_path(paths[i]);
    }

    // load scene's bvh and retrieve mesh bounds
    std::cerr << "loading scene" << std::endl;
    mesh* m = new mesh();
    int numPrimitivesPerLeaf = 0;
    if (!loadBVH("D:\\models\\obj\\staircase.bvh", *m, numPrimitivesPerLeaf)) {
        return -1;
    }
    bbox bounds = m->bounds;
    delete m;
    m = NULL;

    // compute tmax for all paths
    std::cerr << "computing tmax" << std::endl;
    computeTmax(light, numpaths, bounds);

    // compute aabb for all paths
    std::cerr << "computing aabb" << std::endl;
    bbox aabb = computeAABB(light, numpaths);

    // compute Morton code for all rays
    std::cerr << "computing Morton codes" << std::endl;
    MortonKey* keys = new MortonKey[numpaths];
    computeMortonCodes(light, numpaths, aabb, keys);

    // sort rays using their Morton codes
    std::cerr << "sorting paths" << std::endl;
    saved_path* sorted = new saved_path[numpaths];
    sortPaths(paths, numpaths, keys, sorted);

    // save rays to .sorted file
    std::string outfile = filename(bounce, ns, true);
    std::cerr << "saving file " << outfile << std::endl;
    save(outfile, sorted, numpaths);

    //qsort(paths, numpaths, sizeof(saved_path), compareSavedPath);

    //// save rays to .sorted file
    //std::string outfile = filename(bounce, ns, true);
    //std::cerr << "saving file " << outfile << std::endl;
    //save(outfile, paths, numpaths);

    // cleanup
    delete[] paths;
    delete[] light;
    delete[] keys;
    delete[] sorted;

    return 0;
}

void computeTmax(light_path* paths, uint32_t numpaths, bbox bounds) {
    for (auto i = 0; i < numpaths; i++) {
        //tmax_bbox(bounds, paths[i]);
        paths[i].tmax = 1.0f;
    }
}

bbox computeAABB(const light_path* paths, uint32_t numpaths) {
    bbox aabb;
    aabb.min = vec3(FLT_MAX, FLT_MAX, FLT_MAX);
    aabb.max = vec3(-FLT_MAX, -FLT_MAX, -FLT_MAX);

#ifdef DEBUG_SORT
    bool first = true;
#endif
    for (auto i = 0; i < numpaths; i++) {
        const light_path& p = paths[i];
        aabb.min = min(aabb.min, p.origin);
        aabb.max = max(aabb.max, p.origin);

        vec3 far = p.origin + p.tmax * p.direction;
        aabb.min = min(aabb.min, far);
        aabb.max = max(aabb.max, far);
#ifdef DEBUG_SORT
        if (aabb.min[0] == -FLT_MAX && first) {
            std::cerr << "aabb.min = " << aabb.min << std::endl;
            std::cerr << "i = " << i << std::endl;
            std::cerr << "p.origin = " << p.origin << std::endl;
            std::cerr << "p.direction = " << p.direction << std::endl;
            first = false;
        }
#endif
    }
    return aabb;
}

void computeMortonCodes(const light_path* paths, uint32_t numpaths, const bbox aabb, MortonKey* keys) {
    for (auto i = 0; i < numpaths; i++) {
        const light_path& p = paths[i];
        MortonKey& key = keys[i];

        // normalize origin and direction
        vec3 o = (p.origin - aabb.min) / (aabb.max - aabb.min);
        vec3 d = (p.direction - aabb.min) / (aabb.max - aabb.min);

        // generate hash
        key.oldSlot = i;
        for (auto j = 0; j < 6; j++)
            key.hash[j] = 0;
        collectBits(key.hash, 0, (uint32_t)(o[0] * 256.0f * 65536.0f));
        collectBits(key.hash, 1, (uint32_t)(o[1] * 256.0f * 65536.0f));
        collectBits(key.hash, 2, (uint32_t)(o[2] * 256.0f * 65536.0f));
        collectBits(key.hash, 3, (uint32_t)(d[0] * 32.0f * 65536.0f));
        collectBits(key.hash, 4, (uint32_t)(d[1] * 32.0f * 65536.0f));
        collectBits(key.hash, 5, (uint32_t)(d[2] * 32.0f * 65536.0f));
#ifdef DEBUG_SORT
        if (i == 500) {
            std::cerr << "key.oldSlot = " << key.oldSlot << std::endl;
            std::cerr << "key.hash = ";
            for (size_t j = 0; j < 6; j++)
                std::cerr << key.hash[j] << " ";
            std::cerr << std::endl;
            std::cerr << "aabb.min = " << aabb.min << std::endl;
            std::cerr << "aabb.max = " << aabb.max << std::endl;
        }
#endif // DEBUG_SORT
    }
}

void sortPaths(const saved_path* paths, uint32_t numpaths, MortonKey* keys, saved_path* sorted) {
    // sort Morton keys
    qsort(keys, numpaths, sizeof(MortonKey), compareMortonKey);

    // reorder paths using their sort order
    for (auto i = 0; i < numpaths; i++) {
        sorted[i] = paths[keys[i].oldSlot];
    }
}

bool tmax_bbox(const bbox& bounds, light_path& p) {
    float t_min = 0.001f;
    p.tmax = FLT_MAX;
    for (int a = 0; a < 3; a++) {
        float invD = 1.0f / p.direction[a];
        float t0 = (bounds.min[a] - p.origin[a]) * invD;
        float t1 = (bounds.max[a] - p.origin[a]) * invD;
        if (invD < 0.0f) {
            float tmp = t0; t0 = t1; t1 = tmp;
        }
        t_min = t0 > t_min ? t0 : t_min;
        p.tmax = t1 < p.tmax ? t1 : p.tmax;
        if (p.tmax < t_min)
            return false;
    }

    return true;
}

void collectBits(unsigned int* hash, int index, unsigned int value) {
    for (int i = 0; i < 32; i++)
        hash[(index + i * 6) >> 5] |= ((value >> i) & 1) << ((index + i * 6) & 31);
}

int compareMortonKey(const void* A, const void* B) {
    const MortonKey& a = *((const MortonKey*)A);
    const MortonKey& b = *((const MortonKey*)B);
    if (a.hash[5] != b.hash[5]) return (a.hash[5] < b.hash[5]) ? -1 : 1;
    if (a.hash[4] != b.hash[4]) return (a.hash[4] < b.hash[4]) ? -1 : 1;
    if (a.hash[3] != b.hash[3]) return (a.hash[3] < b.hash[3]) ? -1 : 1;
    if (a.hash[2] != b.hash[2]) return (a.hash[2] < b.hash[2]) ? -1 : 1;
    if (a.hash[1] != b.hash[1]) return (a.hash[1] < b.hash[1]) ? -1 : 1;
    if (a.hash[0] != b.hash[0]) return (a.hash[0] < b.hash[0]) ? -1 : 1;
    return 0;
}

int compareSavedPath(const void* A, const void* B) {
    const saved_path& a = *((const saved_path*)A);
    const saved_path& b = *((const saved_path*)B);
    if (a.bitstack == b.bitstack) return 0;
    return a.bitstack < b.bitstack ? -1 : 1;
}
